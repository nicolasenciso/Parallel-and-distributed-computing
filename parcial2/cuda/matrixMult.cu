#include "hip/hip_runtime.h"
 #include <stdio.h>
 #include <stdlib.h>
 #include <assert.h>
 
 #define BLOCK_SIZE 8

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

 int main(int argc, char const *argv[])
 {
     int m, n, k;
     /* Fixed seed for illustration */
     srand(3333);
     n = atoi(argv[1]);
     m = n;
     k = n;
 
     // allocate memory, h_cc is used to store CPU result
     int *h_a, *h_b, *h_c, *h_cc;
     hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
     hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
     hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
     hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);
 
     // random initialize matrix A
     for (int i = 0; i < m; ++i) {
         for (int j = 0; j < n; ++j) {
             h_a[i * n + j] = rand() % 1024;
         }
     }
 
     // random initialize matrix B
     for (int i = 0; i < n; ++i) {
         for (int j = 0; j < k; ++j) {
             h_b[i * k + j] = rand() % 1024;
         }
     }
 
     float gpu_elapsed_time_ms, cpu_elapsed_time_ms;
 
     // some events to count the execution time
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
 
     // start to count execution time of GPU version
     hipEventRecord(start, 0);
     // Allocate memory space on the device 
     int *d_a, *d_b, *d_c;
     hipMalloc((void **) &d_a, sizeof(int)*m*n);
     hipMalloc((void **) &d_b, sizeof(int)*n*k);
     hipMalloc((void **) &d_c, sizeof(int)*m*k);
 
     // copy matrix A and B from host to device memory
     hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
     hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);
 
     unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
     unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
     dim3 dimGrid(grid_cols, grid_rows);
     dim3 dimBlock(BLOCK_SIZE*atoi(argv[2]), BLOCK_SIZE*atoi(argv[2]));
    
     gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

     // Transefr results from device to host 
     hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
     hipDeviceSynchronize();
     // time counting terminate
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
 
     // compute time elapse on GPU computing
     hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    //  printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
 
     // start the CPU version
     hipEventRecord(start, 0);
 
     cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
 
     hipEventRecord(stop, 0);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
     //printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);
 
    
    
    
    printf("%d\t%d\t%f\n",n,atoi(argv[2]), cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    
 
     // free memory
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);
     hipHostFree(h_a);
     hipHostFree(h_b);
     hipHostFree(h_c);
     hipHostFree(h_cc);
     return 0;
 }